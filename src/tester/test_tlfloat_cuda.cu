#include "hip/hip_runtime.h"
// nvcc -O3 --maxrregcount 128 --expt-relaxed-constexpr --std c++20 test_tlfloat_cuda.cu crypt.cpp sha2.cpp

#include <iostream>
#include <chrono>
#include <cmath>
#include <cassert>

#include "tlmath.hpp"
#include "auxiliary.hpp"

using namespace std;
using namespace tlfloat;

int cudaMultiProcessorCount = -1;
int N = 0, blockSize = 0;
int nLoop = 1 << 4, nThread = 1 << 8, nBlock = 0;

#define cudaErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
  if (code != hipSuccess)  {
    fprintf(stderr,"%s(%d) : GPUassert: %s\n", file, line, hipGetErrorString(code));
    if (abort) exit(code);
  }
}

int64_t getTime() {
  return chrono::duration_cast<chrono::microseconds>
    (chrono::high_resolution_clock::now() - chrono::system_clock::from_time_t(0)).count();
}

template<typename T>
T rnd(shared_ptr<RNG> rng) {
  for(;;) {
    T f;
    rng->nextBytes((unsigned char *)&f, sizeof(f));
    f = frexp(f, nullptr);
    if (finite(f)) return f;
  }
}

void *cudaMallocManaged_(size_t size) {
  void *ptr;
  cudaErrorCheck( hipMallocManaged(&ptr, size) );
  return ptr;
}

//

template<typename T>
using Func1 = T (*)(const T&);

template<typename T, Func1<T> func>
__global__ void kern(int n, T *r, T *x) {
  int index = threadIdx.x, stride = blockDim.x;
  for (int i = index; i < n; i += stride) r[i] = func(x[i]);
}

template<typename T, Func1<T> func>
void doTest(const char *mes, T *r, T *x) {
  auto t0 = getTime();
  kern<T, func><<<nBlock, nThread>>>(N, r, x);
  cudaErrorCheck( hipPeekAtLastError() );
  cudaErrorCheck( hipDeviceSynchronize() );

  auto t1 = getTime();
  for (int i = 0; i < N; i++) {
    if (r[i].m == func(x[i]).m) continue;
    cout << mes << " : NG" << endl;
    cout << "arg1   : " << to_string(x[i], 75) << endl;
    cout << "device : " << to_string(r[i], 75) << endl;
    cout << "host   : " << to_string(func(x[i]), 75) << endl;
    exit(-1);
  }
  auto t2 = getTime();
  cout << mes << " : OK (D:" << (t1 - t0)/1000.0 << "ms, H:" << (t2 - t1)/1000.0 << "ms)" << endl;
}

//

template<typename T>
using Func2 = T (*)(const T&, const T&);

template<typename T, Func2<T> func>
__global__ void kern(int n, T *r, T *x, T *y) {
  int index = threadIdx.x, stride = blockDim.x;
  for (int i = index; i < n; i += stride) r[i] = func(x[i], y[i]);
}

template<typename T, Func2<T> func>
void doTest(const char *mes, T *r, T *x, T *y) {
  auto t0 = getTime();
  kern<T, func><<<nBlock, nThread>>>(N, r, x, y);
  cudaErrorCheck( hipPeekAtLastError() );
  cudaErrorCheck( hipDeviceSynchronize() );

  auto t1 = getTime();
  for (int i = 0; i < N; i++) {
    if (r[i].m == func(x[i], y[i]).m) continue;
    cout << mes << " : NG" << endl;
    cout << "arg1   : " << to_string(x[i], 75) << endl;
    cout << "arg2   : " << to_string(y[i], 75) << endl;
    cout << "device : " << to_string(r[i], 75) << endl;
    cout << "host   : " << to_string(func(x[i], y[i]), 75) << endl;
    exit(-1);
  }
  auto t2 = getTime();
  cout << mes << " : OK (D:" << (t1 - t0)/1000.0 << "ms, H:" << (t2 - t1)/1000.0 << "ms)" << endl;
}

//

template<typename T>
using Func3 = T (*)(const T&, const T&, const T&);

template<typename T, Func3<T> func>
__global__ void kern(int n, T *r, T *x, T *y, T *z) {
  int index = threadIdx.x, stride = blockDim.x;
  for (int i = index; i < n; i += stride) r[i] = func(x[i], y[i], z[i]);
}

template<typename T, Func3<T> func>
void doTest(const char *mes, T *r, T *x, T *y, T *z) {
  auto t0 = getTime();
  kern<T, func><<<nBlock, nThread>>>(N, r, x, y, z);
  cudaErrorCheck( hipPeekAtLastError() );
  cudaErrorCheck( hipDeviceSynchronize() );

  auto t1 = getTime();
  for (int i = 0; i < N; i++) {
    if (r[i].m == func(x[i], y[i], z[i]).m) continue;
    cout << mes << " : NG" << endl;
    cout << "arg1   : " << to_string(x[i], 75) << endl;
    cout << "arg2   : " << to_string(y[i], 75) << endl;
    cout << "arg3   : " << to_string(z[i], 75) << endl;
    cout << "device : " << to_string(r[i], 75) << endl;
    cout << "host   : " << to_string(func(x[i], y[i], z[i]), 75) << endl;
    exit(-1);
  }
  auto t2 = getTime();
  cout << mes << " : OK (D:" << (t1 - t0)/1000.0 << "ms, H:" << (t2 - t1)/1000.0 << "ms)" << endl;
}

//

template<typename T> static constexpr T fabs_(const T &a1) { return fabs(a1); }
template<typename T> static constexpr T copysign_(const T &a1, const T &a2) { return copysign(a1, a2); }
template<typename T> static constexpr T fmax_(const T &a1, const T &a2) { return fmax(a1, a2); }
template<typename T> static constexpr T fmin_(const T &a1, const T &a2) { return fmin(a1, a2); }
template<typename T> static constexpr T fdim_(const T &a1, const T &a2) { return fdim(a1, a2); }
template<typename T> static constexpr T add_(const T &a1, const T &a2) { return a1 + a2; }
template<typename T> static constexpr T mul_(const T &a1, const T &a2) { return a1 * a2; }
template<typename T> static constexpr T div_(const T &a1, const T &a2) { return a1 / a2; }
template<typename T> static constexpr T fma_(const T &a1, const T &a2, const T &a3) { return fma(a1, a2, a3); }
template<typename T> static constexpr T sqrt_(const T &a) { return sqrt(a); }
template<typename T> static constexpr T hypot_(const T &a1, const T &a2) { return hypot(a1, a2); }
template<typename T> static constexpr T trunc_(const T &a) { return trunc(a); }
template<typename T> static constexpr T floor_(const T &a) { return floor(a); }
template<typename T> static constexpr T ceil_(const T &a) { return ceil(a); }
template<typename T> static constexpr T round_(const T &a) { return round(a); }
template<typename T> static constexpr T rint_(const T &a) { return rint(a); }
template<typename T> static constexpr T nextafter_(const T &a1, const T &a2) { return nextafter(a1, a2); }

int main(int argc, char **argv) {
  {
    int device;
    cudaErrorCheck(hipGetDevice(&device));

    hipDeviceProp_t devProp;
    cudaErrorCheck(hipGetDeviceProperties(&devProp, device));

    cudaMultiProcessorCount = devProp.multiProcessorCount;
    nBlock = devProp.multiProcessorCount;
  }

  if (argc >= 2) nThread = atoi(argv[1]);
  if (argc >= 3) nLoop = atoi(argv[2]);

  N = nLoop * nThread * nBlock;

  cout << "N = " << N << ", nLoop = " << nLoop << ", nBlock = " << nBlock << ", nThread = " << nThread << endl;

  auto rng = createPreferredRNG();

  Octuple *xo = (Octuple *)cudaMallocManaged_(N*sizeof(Octuple));
  Octuple *yo = (Octuple *)cudaMallocManaged_(N*sizeof(Octuple));
  Octuple *zo = (Octuple *)cudaMallocManaged_(N*sizeof(Octuple));
  Octuple *ro = (Octuple *)cudaMallocManaged_(N*sizeof(Octuple));

  Quad *xq = (Quad *)cudaMallocManaged_(N*sizeof(Quad));
  Quad *yq = (Quad *)cudaMallocManaged_(N*sizeof(Quad));
  Quad *zq = (Quad *)cudaMallocManaged_(N*sizeof(Quad));
  Quad *rq = (Quad *)cudaMallocManaged_(N*sizeof(Quad));

  Double *xd = (Double *)cudaMallocManaged_(N*sizeof(Double));
  Double *yd = (Double *)cudaMallocManaged_(N*sizeof(Double));
  Double *zd = (Double *)cudaMallocManaged_(N*sizeof(Double));
  Double *rd = (Double *)cudaMallocManaged_(N*sizeof(Double));

  Float *xf = (Float *)cudaMallocManaged_(N*sizeof(Float));
  Float *yf = (Float *)cudaMallocManaged_(N*sizeof(Float));
  Float *zf = (Float *)cudaMallocManaged_(N*sizeof(Float));
  Float *rf = (Float *)cudaMallocManaged_(N*sizeof(Float));

  Half *xh = (Half *)cudaMallocManaged_(N*sizeof(Half));
  Half *yh = (Half *)cudaMallocManaged_(N*sizeof(Half));
  Half *zh = (Half *)cudaMallocManaged_(N*sizeof(Half));
  Half *rh = (Half *)cudaMallocManaged_(N*sizeof(Half));

  for (int i = 0; i < N; i++) {
    xo[i] = rnd<Octuple>(rng);
    yo[i] = rnd<Octuple>(rng);
    zo[i] = rnd<Octuple>(rng);

    xq[i] = rnd<Quad>(rng);
    yq[i] = rnd<Quad>(rng);
    zq[i] = rnd<Quad>(rng);

    xd[i] = rnd<Double>(rng);
    yd[i] = rnd<Double>(rng);
    zd[i] = rnd<Double>(rng);

    xf[i] = rnd<Float>(rng);
    yf[i] = rnd<Float>(rng);
    zf[i] = rnd<Float>(rng);

    xh[i] = rnd<Half>(rng);
    yh[i] = rnd<Half>(rng);
    zh[i] = rnd<Half>(rng);
  }

  doTest<Octuple, fabs_>("Octuple fabs", ro, xo);
  doTest<Quad, fabs_>("Quad fabs", rq, xq);
  doTest<Double, fabs_>("Double fabs", rd, xd);
  doTest<Float, fabs_>("Float fabs", rf, xf);
  doTest<Half, fabs_>("Half fabs", rh, xh);

  doTest<Octuple, copysign_>("Octuple copysign", ro, xo, yo);
  doTest<Quad, copysign_>("Quad copysign", rq, xq, yq);
  doTest<Double, copysign_>("Double copysign", rd, xd, yd);
  doTest<Float, copysign_>("Float copysign", rf, xf, yf);
  doTest<Half, copysign_>("Half copysign", rh, xh, yh);

  doTest<Octuple, fmax_>("Octuple fmax", ro, xo, yo);
  doTest<Quad, fmax_>("Quad fmax", rq, xq, yq);
  doTest<Double, fmax_>("Double fmax", rd, xd, yd);
  doTest<Float, fmax_>("Float fmax", rf, xf, yf);
  doTest<Half, fmax_>("Half fmax", rh, xh, yh);

  doTest<Octuple, fmin_>("Octuple fmin", ro, xo, yo);
  doTest<Quad, fmin_>("Quad fmin", rq, xq, yq);
  doTest<Double, fmin_>("Double fmin", rd, xd, yd);
  doTest<Float, fmin_>("Float fmin", rf, xf, yf);
  doTest<Half, fmin_>("Half fmin", rh, xh, yh);

  doTest<Octuple, fdim_>("Octuple fdim", ro, xo, yo);
  doTest<Quad, fdim_>("Quad fdim", rq, xq, yq);
  doTest<Double, fdim_>("Double fdim", rd, xd, yd);
  doTest<Float, fdim_>("Float fdim", rf, xf, yf);
  doTest<Half, fdim_>("Half fdim", rh, xh, yh);

  doTest<Octuple, add_>("Octuple add_", ro, xo, yo);
  doTest<Quad, add_>("Quad add_", rq, xq, yq);
  doTest<Double, add_>("Double add_", rd, xd, yd);
  doTest<Float, add_>("Float add_", rf, xf, yf);
  doTest<Half, add_>("Half add_", rh, xh, yh);

  doTest<Octuple, mul_>("Octuple mul_", ro, xo, yo);
  doTest<Quad, mul_>("Quad mul_", rq, xq, yq);
  doTest<Double, mul_>("Double mul_", rd, xd, yd);
  doTest<Float, mul_>("Float mul_", rf, xf, yf);
  doTest<Half, mul_>("Half mul_", rh, xh, yh);

  doTest<Octuple, div_>("Octuple div_", ro, xo, yo);
  doTest<Quad, div_>("Quad div_", rq, xq, yq);
  doTest<Double, div_>("Double div_", rd, xd, yd);
  doTest<Float, div_>("Float div_", rf, xf, yf);
  doTest<Half, div_>("Half div_", rh, xh, yh);

  doTest<Octuple, fma_>("Octuple fma", ro, xo, yo, zo);
  doTest<Quad, fma_>("Quad fma", rq, xq, yq, zq);
  doTest<Double, fma_>("Double fma", rd, xd, yd, zd);
  doTest<Float, fma_>("Float fma", rf, xf, yf, zf);
  doTest<Half, fma_>("Half fma", rh, xh, yh, zh);

  doTest<Octuple, sqrt_>("Octuple sqrt", ro, xo);
  doTest<Quad, sqrt_>("Quad sqrt", rq, xq);
  doTest<Double, sqrt_>("Double sqrt", rd, xd);
  doTest<Float, sqrt_>("Float sqrt", rf, xf);
  doTest<Half, sqrt_>("Half sqrt", rh, xh);

  doTest<Octuple, hypot_>("Octuple hypot", ro, xo, yo);
  doTest<Quad, hypot_>("Quad hypot", rq, xq, yq);
  doTest<Double, hypot_>("Double hypot", rd, xd, yd);
  doTest<Float, hypot_>("Float hypot", rf, xf, yf);
  doTest<Half, hypot_>("Half hypot", rh, xh, yh);

  doTest<Octuple, trunc_>("Octuple trunc", ro, xo);
  doTest<Quad, trunc_>("Quad trunc", rq, xq);
  doTest<Double, trunc_>("Double trunc", rd, xd);
  doTest<Float, trunc_>("Float trunc", rf, xf);
  doTest<Half, trunc_>("Half trunc", rh, xh);

  doTest<Octuple, floor_>("Octuple floor", ro, xo);
  doTest<Quad, floor_>("Quad floor", rq, xq);
  doTest<Double, floor_>("Double floor", rd, xd);
  doTest<Float, floor_>("Float floor", rf, xf);
  doTest<Half, floor_>("Half floor", rh, xh);

  doTest<Octuple, ceil_>("Octuple ceil", ro, xo);
  doTest<Quad, ceil_>("Quad ceil", rq, xq);
  doTest<Double, ceil_>("Double ceil", rd, xd);
  doTest<Float, ceil_>("Float ceil", rf, xf);
  doTest<Half, ceil_>("Half ceil", rh, xh);

  doTest<Octuple, round_>("Octuple round", ro, xo);
  doTest<Quad, round_>("Quad round", rq, xq);
  doTest<Double, round_>("Double round", rd, xd);
  doTest<Float, round_>("Float round", rf, xf);
  doTest<Half, round_>("Half round", rh, xh);

  doTest<Octuple, rint_>("Octuple rint", ro, xo);
  doTest<Quad, rint_>("Quad rint", rq, xq);
  doTest<Double, rint_>("Double rint", rd, xd);
  doTest<Float, rint_>("Float rint", rf, xf);
  doTest<Half, rint_>("Half rint", rh, xh);

  doTest<Octuple, nextafter_>("Octuple nextafter", ro, xo, yo);
  doTest<Quad, nextafter_>("Quad nextafter", rq, xq, yq);
  doTest<Double, nextafter_>("Double nextafter", rd, xd, yd);
  doTest<Float, nextafter_>("Float nextafter", rf, xf, yf);
  doTest<Half, nextafter_>("Half nextafter", rh, xh, yh);

  doTest<Octuple, sin>("Octuple sin", ro, xo);
  doTest<Quad, sin>("Quad sin", rq, xq);
  doTest<Double, sin>("Double sin", rd, xd);
  doTest<Float, sin>("Float sin", rf, xf);
  doTest<Half, sin>("Half sin", rh, xh);

  doTest<Octuple, cos>("Octuple cos", ro, xo);
  doTest<Quad, cos>("Quad cos", rq, xq);
  doTest<Double, cos>("Double cos", rd, xd);
  doTest<Float, cos>("Float cos", rf, xf);
  doTest<Half, cos>("Half cos", rh, xh);

  doTest<Octuple, tan>("Octuple tan", ro, xo);
  doTest<Quad, tan>("Quad tan", rq, xq);
  doTest<Double, tan>("Double tan", rd, xd);
  doTest<Float, tan>("Float tan", rf, xf);
  doTest<Half, tan>("Half tan", rh, xh);

  doTest<Octuple, atan2>("Octuple atan2", ro, xo, yo);
  doTest<Quad, atan2>("Quad atan2", rq, xq, yq);
  doTest<Double, atan2>("Double atan2", rd, xd, yd);
  doTest<Float, atan2>("Float atan2", rf, xf, yf);
  doTest<Half, atan2>("Half atan2", rh, xh, yh);

  doTest<Octuple, asin>("Octuple asin", ro, xo);
  doTest<Quad, asin>("Quad asin", rq, xq);
  doTest<Double, asin>("Double asin", rd, xd);
  doTest<Float, asin>("Float asin", rf, xf);
  doTest<Half, asin>("Half asin", rh, xh);

  doTest<Octuple, acos>("Octuple acos", ro, xo);
  doTest<Quad, acos>("Quad acos", rq, xq);
  doTest<Double, acos>("Double acos", rd, xd);
  doTest<Float, acos>("Float acos", rf, xf);
  doTest<Half, acos>("Half acos", rh, xh);

  doTest<Octuple, atan>("Octuple atan", ro, xo);
  doTest<Quad, atan>("Quad atan", rq, xq);
  doTest<Double, atan>("Double atan", rd, xd);
  doTest<Float, atan>("Float atan", rf, xf);
  doTest<Half, atan>("Half atan", rh, xh);

  doTest<Octuple, log>("Octuple log", ro, xo);
  doTest<Quad, log>("Quad log", rq, xq);
  doTest<Double, log>("Double log", rd, xd);
  doTest<Float, log>("Float log", rf, xf);
  doTest<Half, log>("Half log", rh, xh);

  doTest<Octuple, log2>("Octuple log2", ro, xo);
  doTest<Quad, log2>("Quad log2", rq, xq);
  doTest<Double, log2>("Double log2", rd, xd);
  doTest<Float, log2>("Float log2", rf, xf);
  doTest<Half, log2>("Half log2", rh, xh);

  doTest<Octuple, log10>("Octuple log10", ro, xo);
  doTest<Quad, log10>("Quad log10", rq, xq);
  doTest<Double, log10>("Double log10", rd, xd);
  doTest<Float, log10>("Float log10", rf, xf);
  doTest<Half, log10>("Half log10", rh, xh);

  doTest<Octuple, log1p>("Octuple log1p", ro, xo);
  doTest<Quad, log1p>("Quad log1p", rq, xq);
  doTest<Double, log1p>("Double log1p", rd, xd);
  doTest<Float, log1p>("Float log1p", rf, xf);
  doTest<Half, log1p>("Half log1p", rh, xh);

  doTest<Octuple, exp>("Octuple exp", ro, xo);
  doTest<Quad, exp>("Quad exp", rq, xq);
  doTest<Double, exp>("Double exp", rd, xd);
  doTest<Float, exp>("Float exp", rf, xf);
  doTest<Half, exp>("Half exp", rh, xh);

  doTest<Octuple, exp2>("Octuple exp2", ro, xo);
  doTest<Quad, exp2>("Quad exp2", rq, xq);
  doTest<Double, exp2>("Double exp2", rd, xd);
  doTest<Float, exp2>("Float exp2", rf, xf);
  doTest<Half, exp2>("Half exp2", rh, xh);

  doTest<Octuple, exp10>("Octuple exp10", ro, xo);
  doTest<Quad, exp10>("Quad exp10", rq, xq);
  doTest<Double, exp10>("Double exp10", rd, xd);
  doTest<Float, exp10>("Float exp10", rf, xf);
  doTest<Half, exp10>("Half exp10", rh, xh);

  doTest<Octuple, expm1>("Octuple expm1", ro, xo);
  doTest<Quad, expm1>("Quad expm1", rq, xq);
  doTest<Double, expm1>("Double expm1", rd, xd);
  doTest<Float, expm1>("Float expm1", rf, xf);
  doTest<Half, expm1>("Half expm1", rh, xh);

  doTest<Octuple, pow>("Octuple pow", ro, xo, yo);
  doTest<Quad, pow>("Quad pow", rq, xq, yq);
  doTest<Double, pow>("Double pow", rd, xd, yd);
  doTest<Float, pow>("Float pow", rf, xf, yf);
  doTest<Half, pow>("Half pow", rh, xh, yh);

  doTest<Octuple, cbrt>("Octuple cbrt", ro, xo);
  doTest<Quad, cbrt>("Quad cbrt", rq, xq);
  doTest<Double, cbrt>("Double cbrt", rd, xd);
  doTest<Float, cbrt>("Float cbrt", rf, xf);
  doTest<Half, cbrt>("Half cbrt", rh, xh);

  doTest<Octuple, sinh>("Octuple sinh", ro, xo);
  doTest<Quad, sinh>("Quad sinh", rq, xq);
  doTest<Double, sinh>("Double sinh", rd, xd);
  doTest<Float, sinh>("Float sinh", rf, xf);
  doTest<Half, sinh>("Half sinh", rh, xh);

  doTest<Octuple, cosh>("Octuple cosh", ro, xo);
  doTest<Quad, cosh>("Quad cosh", rq, xq);
  doTest<Double, cosh>("Double cosh", rd, xd);
  doTest<Float, cosh>("Float cosh", rf, xf);
  doTest<Half, cosh>("Half cosh", rh, xh);

  doTest<Octuple, tanh>("Octuple tanh", ro, xo);
  doTest<Quad, tanh>("Quad tanh", rq, xq);
  doTest<Double, tanh>("Double tanh", rd, xd);
  doTest<Float, tanh>("Float tanh", rf, xf);
  doTest<Half, tanh>("Half tanh", rh, xh);

  doTest<Octuple, asinh>("Octuple asinh", ro, xo);
  doTest<Quad, asinh>("Quad asinh", rq, xq);
  doTest<Double, asinh>("Double asinh", rd, xd);
  doTest<Float, asinh>("Float asinh", rf, xf);
  doTest<Half, asinh>("Half asinh", rh, xh);

  doTest<Octuple, acosh>("Octuple acosh", ro, xo);
  doTest<Quad, acosh>("Quad acosh", rq, xq);
  doTest<Double, acosh>("Double acosh", rd, xd);
  doTest<Float, acosh>("Float acosh", rf, xf);
  doTest<Half, acosh>("Half acosh", rh, xh);

  doTest<Octuple, atanh>("Octuple atanh", ro, xo);
  doTest<Quad, atanh>("Quad atanh", rq, xq);
  doTest<Double, atanh>("Double atanh", rd, xd);
  doTest<Float, atanh>("Float atanh", rf, xf);
  doTest<Half, atanh>("Half atanh", rh, xh);

  doTest<Octuple, fmod>("Octuple fmod", ro, xo, yo);
  doTest<Quad, fmod>("Quad fmod", rq, xq, yq);
  doTest<Double, fmod>("Double fmod", rd, xd, yd);
  doTest<Float, fmod>("Float fmod", rf, xf, yf);
  doTest<Half, fmod>("Half fmod", rh, xh, yh);

  doTest<Octuple, remainder>("Octuple remainder", ro, xo, yo);
  doTest<Quad, remainder>("Quad remainder", rq, xq, yq);
  doTest<Double, remainder>("Double remainder", rd, xd, yd);
  doTest<Float, remainder>("Float remainder", rf, xf, yf);
  doTest<Half, remainder>("Half remainder", rh, xh, yh);

  hipFree(rh);
  hipFree(zh);
  hipFree(yh);
  hipFree(xh);

  hipFree(rf);
  hipFree(zf);
  hipFree(yf);
  hipFree(xf);

  hipFree(rd);
  hipFree(zd);
  hipFree(yd);
  hipFree(xd);

  hipFree(rq);
  hipFree(zq);
  hipFree(yq);
  hipFree(xq);

  hipFree(ro);
  hipFree(zo);
  hipFree(yo);
  hipFree(xo);

  return 0;
}
